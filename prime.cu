#include "hip/hip_runtime.h"
// Elapsed Real Time for input-4.txt:
// Type of GPU:

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Resizable array for a list of values to test for prime-ness
int *vList;

// Number of values on the list.
int vCount = 0;

// Capacity of the list of values.
int vCap = 0;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: prime [report]\n" );
  exit( 1 );
}

// Read the list of numbers to test for prime-ness
void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value.
    vList[ vCount++ ] = v;
  }
}

// Check my assigned element of vList to see if it's prime (you'll need to add
// some parameters)
__global__ void checkPrime( int vCount,
                            bool report,
                            int* values,
                            int* results ) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if ( idx < vCount ) {
    bool prime = true;
    int value = values[idx];
    if(value % 2 == 0) {
      prime = false;
    } else {
      for(int i = 3; i * i <= value; i +=2 ){
        if(value % i == 0){
          prime = false;
          break;
        }
      }
    }
    if(prime){
      fprintf(stdout, "%d\n", value);
      results[idx] = 1;
    } else{
      results[idx] = 0;
    }
  }
}

int main( int argc, char *argv[] ) {
  if ( argc < 1 || argc > 2 )
    usage();

  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();

  // Add code to allocate memory on the device and copy over the list.
  int *valueList = NULL;
  if ( hipMalloc((void **)&valueList, vCount * sizeof(int) ) != hipSuccess )
    fail( "Failed to allocate space for lenght list on device" );

  // Copy the list over to the device.
  if ( hipMemcpy( valueList, vList, vCount * sizeof(int),
                   hipMemcpyHostToDevice) != hipSuccess )
    fail( "Failed to copy list to device" );

  // Add code to allocate space on the device to hold the results.
  int *resultsList = NULL;
  resultsList = (int *) malloc( vCount * sizeof( int ) );
  if ( hipMalloc((void **)&resultsList, vCount * sizeof(int) ) != hipSuccess )
    fail( "Failed to allocate space for lenght list on device" );

  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions (you'll need to add some parameters)
  checkPrime<<<blocksPerGrid, threadsPerBlock>>>( vCount, report, *valueList, *resultsList );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // Add code to copy results back to the host then add up the total number
  // of primes found.
  
  int* results = malloc(vCount * sizeof(int));

  if ( hipMemcpy( results, resultsList, vCount * sizeof(int),
                   hipMemcpyDeviceToHost) != hipSuccess )
    fail( "Can't copy list from device to host" );
  
  int primeCount = 0;
  for(int i = 0; i < vCount; i++) {
    if(results[i] == 1) {
      primeCount++;
    }
  }

  fprintf(stdout, "Prime count: %d\n", primeCount);

  // Free memory on the device and the host.
  hipFree(resultsList);
  hipFree(valueList);
  free( results )
  free( vList );

  hipDeviceReset();

  return 0;
}
